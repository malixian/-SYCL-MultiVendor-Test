#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

const size_t REPEAT = 100;

const size_t ITER = 10000;

#define CHECK(call) \
{ \
    const hipError_t error = call; \
    if (error != hipSuccess) { \
        std::cerr << "Error: " << __FILE__ << ", line " << __LINE__ << ": " \
                  << hipGetErrorString(error) << std::endl; \
        exit(1); \
    } \
}

// 向量加法的CUDA核函数
__global__ void addVectors(float* c, const float* a, const float* b, int n) {
    for(int rid=0; rid<ITER; rid++) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < n) {
            c[i] = a[i] + b[i];
        }
    }
    
    
    
}

int main() {
    const int n = 102400; // 向量长度
    float h_a[n], h_b[n], h_c[n]; // 主机端向量（h_表示host）
    float *d_a, *d_b, *d_c; // 设备端向量（d_表示device）

    // 初始化主机端向量
    for (int i = 0; i < n; ++i) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }
    
    // 分配GPU内存
    CHECK(hipMalloc((void**)&d_a, n * sizeof(float)));
    CHECK(hipMalloc((void**)&d_b, n * sizeof(float)));
    CHECK(hipMalloc((void**)&d_c, n * sizeof(float)));

    //auto start_time = std::chrono::steady_clock::now();
    // 将数据从主机复制到设备
    CHECK(hipMemcpy(d_a, h_a, n * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, h_b, n * sizeof(float), hipMemcpyHostToDevice));

    // 配置并启动核函数
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    
    auto start_time = std::chrono::steady_clock::now();
    
    for(int i=0; i<REPEAT; i++)
        addVectors<<<numBlocks, blockSize>>>(d_c, d_a, d_b, n);

    // 同步以确保核函数执行完毕
    CHECK(hipDeviceSynchronize());
    
    auto end_time = std::chrono::steady_clock::now();
    auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end_time - start_time).count();
    printf("Total kernel execution time %f (ms)\n", time / REPEAT * 1e-6f);

    // 将结果从设备复制回主机
    CHECK(hipMemcpy(h_c, d_c, n * sizeof(float), hipMemcpyDeviceToHost));

    // 释放GPU内存
    CHECK(hipFree(d_a));
    CHECK(hipFree(d_b));
    CHECK(hipFree(d_c));

    // 打印结果
    std::cout << "Check PASS:\n";

    return 0;
}
