#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

const int N = 102400; // 向量维度
const size_t REPEAT = 100;
const size_t ITER = 10000;

#define CHECK(call) \
{ \
    const hipError_t error = call; \
    if (error != hipSuccess) { \
        std::cerr << "Error: " << __FILE__ << ", line " << __LINE__ << ": " \
                  << hipGetErrorString(error) << std::endl; \
        exit(1); \
    } \
}

__global__ void batchnorm(float *x, float *mean, float *variance, float *scale, float *bias, float *output)
{
    for(int rid=0; rid<ITER; rid++) {
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        if (tid < N)
        {
            output[tid] = scale[tid] * (x[tid] - mean[tid]) / sqrtf(variance[tid] + 1e-5) + bias[tid];
        }
    }
}

int main()
{

    float h_x[N], h_mean[N], h_variance[N], h_scale[N], h_bias[N], h_output[N]; // 主机端向量（h_表示host）
    float *x, *mean, *variance, *scale, *bias, *output;

    for (int i = 0; i < N; ++i)
    {
        h_x[i] = i;
        h_mean[i] = i * 2;
        h_variance[i] = i * 3;
        h_scale[i] = 1.0;
        h_bias[i] = 0.0;
    
    }
    
    

    CHECK(hipMalloc((void**)&x, N * sizeof(float)));
    CHECK(hipMalloc((void**)&mean, N * sizeof(float)));
    CHECK(hipMalloc((void**)&variance, N * sizeof(float)));
    CHECK(hipMalloc((void**)&scale, N * sizeof(float)));
    CHECK(hipMalloc((void**)&bias, N * sizeof(float)));
    CHECK(hipMalloc((void**)&output, N * sizeof(float)));

    

    // 将数据从主机复制到设备
    CHECK(hipMemcpy(x, h_x, N * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(mean, h_mean, N * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(variance, h_variance, N * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(scale, h_scale, N * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(bias, h_bias, N * sizeof(float), hipMemcpyHostToDevice));

    dim3 threadsPerBlock(256);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    auto start_time = std::chrono::steady_clock::now();
    float kernel_milliseconds = 0;

    for(int r=0; r<REPEAT; r++)
    batchnorm<<<numBlocks, threadsPerBlock>>>(x, mean, variance, scale, bias, output);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    auto end_time = std::chrono::steady_clock::now();
    auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end_time - start_time).count();
    printf("Total kernel execution time %f (ms)\n", time / REPEAT * 1e-6f);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    kernel_milliseconds += milliseconds;

    hipFree(x);
    hipFree(mean);
    hipFree(variance);
    hipFree(scale);
    hipFree(bias);
    hipFree(output);
    
    
    

    return 0;
}

