
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>

const size_t width = 10240;
const size_t height = 10240;
const int kernelSize = 3;
const int REPEAT = 100;

__global__ void conv2d(float *output, float *input, float *kernel)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < height && col < width) {
        int sum = 0;
        for (int kRow = 0; kRow < kernelSize; ++kRow) {
            for (int kCol = 0; kCol < kernelSize; ++kCol) {
                int inputRow = row + kRow - kernelSize / 2;
                int inputCol = col + kCol - kernelSize / 2;
                if (inputRow >= 0 && inputRow < height && inputCol >= 0 && inputCol < width)
                    sum += input[inputRow * width + inputCol] * kernel[kRow * kernelSize + kCol];
            }
        }
        output[row * width + col] = sum;
    }
}

int main()
{
    // 创建输入、输出和卷积核数据
    std::vector<float> input_data(width * height, 1.0f);
    std::vector<float> kernel_data(kernelSize * kernelSize, 1.0f);
    std::vector<float> output_data(width * height);

    // 将数据从主机内存拷贝到设备内存
    float *d_input, *d_kernel, *d_output;
    hipMalloc(&d_input, width * height * sizeof(float));
    hipMalloc(&d_kernel, kernelSize * kernelSize * sizeof(float));
    hipMalloc(&d_output, width * height * sizeof(float));

    hipMemcpy(d_input, input_data.data(), width * height * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel_data.data(), kernelSize * kernelSize * sizeof(float), hipMemcpyHostToDevice);

    auto start_time = std::chrono::steady_clock::now();

    // 定义 CUDA 线程块和网格大小
    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // 调用 CUDA kernel 进行卷积操作
    for(int rid=0; rid<REPEAT; rid++)
        conv2d<<<gridSize, blockSize>>>(d_output, d_input, d_kernel);
    hipDeviceSynchronize();

    auto end_time = std::chrono::steady_clock::now();
    auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end_time - start_time).count();
    printf("Total kernel execution time %f (ms)\n", time / REPEAT * 1e-6f);

    // 将结果从设备内存拷贝回主机内存
    hipMemcpy(output_data.data(), d_output, width * height * sizeof(float), hipMemcpyDeviceToHost);

    // 释放设备内存
    hipFree(d_input);
    hipFree(d_kernel);
    hipFree(d_output);

    return 0;
}

