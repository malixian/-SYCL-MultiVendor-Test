
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>


const size_t matrixSize = 1024;
const int REPEAT = 100;


__global__ void matrixMultiplication(float *matrixA, float *matrixB, float *result)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0f;
    for (int i = 0; i < matrixSize; ++i) {
        sum += matrixA[row * matrixSize + i] * matrixB[i * matrixSize + col];
    }
    result[row * matrixSize + col] = sum;
    
}

int main() {
    std::vector<float> matrixA(matrixSize * matrixSize, 1.0f);
    std::vector<float> matrixB(matrixSize * matrixSize, 1.0f);
    std::vector<float> result(matrixSize * matrixSize);

    float *d_matrixA, *d_matrixB, *d_result;
    hipMalloc(&d_matrixA, matrixSize * matrixSize * sizeof(float));
    hipMalloc(&d_matrixB, matrixSize * matrixSize * sizeof(float));
    hipMalloc(&d_result, matrixSize * matrixSize * sizeof(float));

    auto start_time = std::chrono::steady_clock::now();
    hipMemcpy(d_matrixA, matrixA.data(), matrixSize * matrixSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_matrixB, matrixB.data(), matrixSize * matrixSize * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((matrixSize + blockSize.x - 1) / blockSize.x, (matrixSize + blockSize.y - 1) / blockSize.y);

    
    for(int rid=0; rid<REPEAT; rid++)
        matrixMultiplication<<<gridSize, blockSize>>>(d_matrixA, d_matrixB, d_result);
    hipDeviceSynchronize();

    auto end_time = std::chrono::steady_clock::now();
    auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end_time - start_time).count();
    printf("Total kernel execution time %f (ms)\n", time / REPEAT * 1e-6f);

    hipMemcpy(result.data(), d_result, matrixSize * matrixSize * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_matrixA);
    hipFree(d_matrixB);
    hipFree(d_result);

    return 0;
}

